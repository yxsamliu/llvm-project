// REQUIRES: amdgpu-registered-target

// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa -target-cpu gfx906 \
// RUN:   -aux-triple x86_64-unknown-linux-gnu -fcuda-is-device \
// RUN:   -emit-llvm -o - -x hip %s -debug-info-kind=limited \
// RUN:   | FileCheck %s

// Check no assertion with debug info.

// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa -target-cpu gfx906 \
// RUN:   -aux-triple x86_64-unknown-linux-gnu -fcuda-is-device \
// RUN:   -S -o %t.s -x hip %s \
// RUN:   -debug-info-kind=limited

#include "Inputs/hip/hip_runtime.h"
 
struct A {
  int x[100];
  __device__ A();
};

struct B {
  int x[100];
};

__device__ B b;

__device__ void callee(A *a);

// CHECK-LABEL: @_Z5func1v(
// CHECK-SAME: %struct.A addrspace(5)* noalias sret(%struct.A) align 4 %[[RET:.*]])
// CHECK: %x = alloca [100 x i32], align 16, addrspace(5)
// CHECK: %x.ascast = addrspacecast [100 x i32] addrspace(5)* %x to [100 x i32]*
// CHECK: %p = alloca %struct.A*, align 8, addrspace(5)
// CHECK: %p.ascast = addrspacecast %struct.A* addrspace(5)* %p to %struct.A**
// CHECK: %[[RET_CAST:.*]] = addrspacecast %struct.A addrspace(5)* %[[RET]] to %struct.A*
// CHECK: call void @llvm.dbg.declare(metadata %struct.A addrspace(5)* %[[RET]]
// CHECK: call void @_ZN1AC1Ev(%struct.A* nonnull dereferenceable(400) %[[RET_CAST]])
// CHECK: call void @llvm.dbg.declare(metadata [100 x i32] addrspace(5)* %x
// CHECK: call void @_Z6calleeP1A(%struct.A* %[[RET_CAST]])
// CHECK: %[[RET_CAST2:.*]] = bitcast %struct.A* %[[RET_CAST]] to i8*
// CHECK: call void @llvm.memcpy.p0i8.p0i8.i64(i8* align 4 %[[RET_CAST2]], i8* align 16 %{{.*}}, i64 400, i1 false)
// CHECK: call void @llvm.dbg.declare(metadata %struct.A* addrspace(5)* %p
// CHECK: store %struct.A* %[[RET_CAST]], %struct.A** %p.ascast
__device__ A func1() {
  A a;
  int x[100];
  callee(&a);
  __builtin_memcpy(&a, x, 400);
  A *p = &a;
  return a;
}

// CHECK-LABEL: @_Z6func1av(%struct.B addrspace(5)* noalias sret(%struct.B) align 4 
__device__ B func1a() {
  B b;
  return b;
}

// Check returning the return value again.

// CHECK-LABEL: @_Z5func2v(
// CHECK-SAME: %struct.A addrspace(5)* noalias sret(%struct.A) align 4 %[[RET:.*]])
// CHECK: %[[CAST1:.*]] = addrspacecast %struct.A addrspace(5)* %[[RET]] to %struct.A*
// CHECK: %[[CAST2:.*]] = addrspacecast %struct.A* %[[CAST1]] to %struct.A addrspace(5)*
// CHECK: call void @_Z5func1v(%struct.A addrspace(5)* sret(%struct.A) align 4 %[[CAST2]])
__device__ A func2() {
  A a = func1();
  return a;
}

// Check assigning the return value to a global variable.

// CHECK-LABEL: @_Z5func3v(
// CHECK: %[[RET:.*]] = alloca %struct.B, align 4, addrspace(5)
// CHECK: %[[CAST1:.*]] = addrspacecast %struct.B addrspace(5)* %[[RET]] to %struct.B*
// CHECK: %[[CAST2:.*]] = addrspacecast %struct.B* %[[CAST1]] to %struct.B addrspace(5)*
// CHECK: call void @_Z6func1av(%struct.B addrspace(5)* sret(%struct.B) align 4 %[[CAST2]]
// CHECK: %[[CAST3:.*]] = bitcast %struct.B* %[[CAST1]] to i8*
// CHECK: call void @llvm.memcpy.p0i8.p0i8.i64{{.*}}@b{{.*}}%[[CAST3]]
__device__ void func3() {
  b = func1a();
}

// Check assigning the return value to a temporary variable.

// CHECK-LABEL: @_Z5func4v(
// CHECK: %[[TMP:.*]] = alloca %struct.A, align 4, addrspace(5)
// CHECK: %[[TMP_CAST1:.*]] = addrspacecast %struct.A addrspace(5)* %[[TMP]] to %struct.A*
// CHECK: %[[RET:.*]] = alloca %struct.A, align 4, addrspace(5)
// CHECK: %[[RET_CAST1:.*]] = addrspacecast %struct.A addrspace(5)* %[[RET]] to %struct.A*
// CHECK: call void @_ZN1AC1Ev(%struct.A* nonnull dereferenceable(400) %[[TMP_CAST1]])
// CHECK: %[[RET_CAST2:.*]] = addrspacecast %struct.A* %[[RET_CAST1]] to %struct.A addrspace(5)*
// CHECK: call void @_Z5func1v(%struct.A addrspace(5)* sret(%struct.A) align 4 %[[RET_CAST2]]
// CHECK: %[[TMP_CAST2:.*]] = bitcast %struct.A* %[[TMP_CAST1]] to i8*
// CHECK: %[[RET_CAST3:.*]] = bitcast %struct.A* %[[RET_CAST1]] to i8*
// CHECK: call void @llvm.memcpy.p0i8.p0i8.i64{{.*}}%[[TMP_CAST2]]{{.*}}%[[RET_CAST3]]
__device__ void func4() {
  A a;
  a = func1();
}
