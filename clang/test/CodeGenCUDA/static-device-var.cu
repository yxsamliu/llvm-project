#include "hip/hip_runtime.h"
// REQUIRES: x86-registered-target
// REQUIRES: amdgpu-registered-target

// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa -fcuda-is-device \
// RUN:   -emit-llvm -o - -x hip %s | FileCheck \
// RUN:   -check-prefixes=DEV,INT-DEV %s

// RUN: %clang_cc1 -triple x86_64-gnu-linux \
// RUN:   -emit-llvm -o - -x hip %s | FileCheck \
// RUN:   -check-prefixes=HOST,INT-HOST %s

// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa -fcuda-is-device -cuid=123abc \
// RUN:   -emit-llvm -o - -x hip %s | FileCheck \
// RUN:   -check-prefixes=DEV,EXT-DEV %s

// RUN: %clang_cc1 -triple x86_64-gnu-linux -cuid=123abc \
// RUN:   -emit-llvm -o - -x hip %s | FileCheck \
// RUN:   -check-prefixes=HOST,EXT-HOST %s

#include "Inputs/hip/hip_runtime.h"

// Test function scope static device variable, which should not be externalized.
// DEV-DAG: @_ZZ6kernelPiPPKiE1w = internal addrspace(4) constant i32 1

// Test normal static device variables
// INT-DEV-DAG: @_ZL1x = internal addrspace(1) global i32 0
// INT-HOST-DAG: @_ZL1x = internal global i32 undef
// INT-HOST-DAG: @[[DEVNAMEX:[0-9]+]] = {{.*}}c"_ZL1x\00"

// Test externalized static device variables
// EXT-DEV-DAG: @_ZL1x.static.123abc = addrspace(1) externally_initialized global i32 0
// EXT-HOST-DAG: @_ZL1x.static.123abc = internal global i32 undef
// EXT-HOST-DAG: @[[DEVNAMEX:[0-9]+]] = {{.*}}c"_ZL1x.static.123abc\00"

static __device__ int x;

// Test normal static device variables
// INT-DEV-DAG: @_ZL1y = internal addrspace(4) global i32 0
// INT-HOST-DAG: @_ZL1y = internal global i32 undef
// INT-HOST-DAG: @[[DEVNAMEY:[0-9]+]] = {{.*}}c"_ZL1y\00"

// Test externalized static device variables
// EXT-DEV-DAG: @_ZL1y.static.123abc = addrspace(4) externally_initialized global i32 0
// EXT-HOST-DAG: @_ZL1y.static.123abc = internal global i32 undef
// EXT-HOST-DAG: @[[DEVNAMEY:[0-9]+]] = {{.*}}c"_ZL1y.static.123abc\00"

static __constant__ int y;

// Test static host variable, which should not be externalized nor registered.
// HOST-DAG: @_ZL1z = internal global i32 0
// DEV-NOT: @_ZL1z
static int z;

// Test static device variable in inline function, which should not be
// externalized nor registered.
// DEV-DAG: @_ZZ6devfunPPKiE1p = linkonce_odr addrspace(4) constant i32 2, comdat

inline __device__ void devfun(const int ** b) {
  const static int p = 2;
  b[0] = &p;
}

__global__ void kernel(int *a, const int **b) {
  const static int w = 1;
  a[0] = x;
  a[1] = y;
  b[0] = &w;
  devfun(b);
}

int* getDeviceSymbol(int *x);

void foo() {
  getDeviceSymbol(&x);
  getDeviceSymbol(&y);
  z = 123;
}

// INT-HOST: __hipRegisterVar({{.*}}@_ZL1x{{.*}}@[[DEVNAMEX]]
// INT-HOST: __hipRegisterVar({{.*}}@_ZL1y{{.*}}@[[DEVNAMEY]]
// EXT-HOST: __hipRegisterVar({{.*}}@_ZL1x.static.123abc{{.*}}@[[DEVNAMEX]]
// EXT-HOST: __hipRegisterVar({{.*}}@_ZL1y.static.123abc{{.*}}@[[DEVNAMEY]]
// HOST-NOT: __hipRegisterVar({{.*}}@_ZZ6kernelPiPPKiE1w
// HOST-NOT: __hipRegisterVar({{.*}}@_ZZ6devfunPPKiE1p
