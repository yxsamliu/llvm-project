// RUN: %clang_cc1 -std=c++11 -triple amdgcn-amd-amdhsa \
// RUN:   -fcuda-is-device -emit-llvm -o - -x hip %s | FileCheck %s
// RUN: %clang_cc1 -std=c++11 -triple amdgcn-amd-amdhsa \
// RUN:   -fcuda-is-device -emit-llvm -o - -x hip %s | FileCheck -check-prefix=NEG %s

#include "Inputs/hip/hip_runtime.h"

struct A {
  int x;
};

constexpr int constexpr_var = 1;
constexpr A constexpr_struct{2};
constexpr A constexpr_array[4] = {0, 0, 0, 3};
constexpr char constexpr_str[] = "abcd";
const int const_var = 4;
const A const_struct{5};
const A const_array[] = {0, 0, 0, 6};
const char const_str[] = "xyz";

// CHECK-DAG: @_ZL13constexpr_str.const = private unnamed_addr addrspace(4) constant [5 x i8] c"abcd\00"
// CHECK-DAG: @_ZL13constexpr_var = internal addrspace(4) constant i32 1
// CHECK-DAG: @_ZL16constexpr_struct = internal addrspace(4) constant %struct.A { i32 2 }
// CHECK-DAG: @_ZL15constexpr_array = internal addrspace(4) constant [4 x %struct.A] [%struct.A zeroinitializer, %struct.A zeroinitializer, %struct.A zeroinitializer, %struct.A { i32 3 }]
// CHECK-DAG: @_ZL9const_var = internal addrspace(4) constant i32 4
// CHECK-DAG: @_ZL12const_struct = internal addrspace(4) constant %struct.A { i32 5 }
// CHECK-DAG: @_ZL11const_array = internal addrspace(4) constant [4 x %struct.A] [%struct.A zeroinitializer, %struct.A zeroinitializer, %struct.A zeroinitializer, %struct.A { i32 6 }]
// CHECK-DAG: @_ZL9const_str = internal addrspace(4) constant [4 x i8] c"xyz\00"

// NEG-NOT: external

// CHECK-LABEL: define{{.*}}@_Z7dev_funPiPPKi
// CHECK: store i32 1
// CHECK: store i32 2
// CHECK: store i32 3
// CHECK: load i8, i8* getelementptr {{.*}} @_ZL13constexpr_str.const
// CHECK: store i32 4
// CHECK: store i32 5
// CHECK: store i32 6
// CHECK: load i8, i8* getelementptr {{.*}} @_ZL9const_str
// CHECK: store i32* {{.*}}@_ZL13constexpr_var
// CHECK: store i32* getelementptr {{.*}} @_ZL16constexpr_struct
// CHECK: store i32* getelementptr {{.*}} @_ZL15constexpr_array
// CHECK: store i32* {{.*}}@_ZL9const_var
// CHECK: store i32* getelementptr {{.*}} @_ZL12const_struct
// CHECK: store i32* getelementptr {{.*}} @_ZL11const_array
__device__ void dev_fun(int *out, const int **out2) {
  *out = constexpr_var;
  *out = constexpr_struct.x;
  *out = constexpr_array[3].x;
  *out = constexpr_str[3];
  *out = const_var;
  *out = const_struct.x;
  *out = const_array[3].x;
  *out = const_str[3];
  *out2 = &constexpr_var;
  *out2 = &constexpr_struct.x;
  *out2 = &constexpr_array[3].x;
  *out2 = &const_var;
  *out2 = &const_struct.x;
  *out2 = &const_array[3].x;
}
