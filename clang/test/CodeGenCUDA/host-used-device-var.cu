#include "hip/hip_runtime.h"
// REQUIRES: amdgpu-registered-target
// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa -fcuda-is-device -x hip %s \
// RUN:   -std=c++17 -O3 -mllvm -amdgpu-internalize-symbols -emit-llvm -o - \
// RUN:   | FileCheck -check-prefix=DEV %s
// RUN: %clang_cc1 -triple x86_64-unknown-linux-gnu -x hip %s \
// RUN:   -std=c++17 -O3 -emit-llvm -o - | FileCheck -check-prefix=HOST %s

// Negative tests.

// RUN: %clang_cc1 -triple amdgcn-amd-amdhsa -fcuda-is-device -x hip %s \
// RUN:   -std=c++17 -O3 -mllvm -amdgpu-internalize-symbols -emit-llvm -o - \
// RUN:   | FileCheck -check-prefix=DEV-NEG %s
// RUN: %clang_cc1 -triple x86_64-unknown-linux-gnu -x hip %s \
// RUN:   -std=c++17 -O3 -emit-llvm -o - | FileCheck -check-prefix=HOST-NEG %s

#include "Inputs/hip/hip_runtime.h"

// Check device variables used by neither host nor device functioins are not kept.

// DEV-NEG-NOT: @v1
__device__ int v1;

// DEV-NEG-NOT: @v2
__constant__ int v2;

// DEV-NEG-NOT: @_ZL2v3
static __device__ int v3;

// Check device variables used by host functions are kept.

// DEV-DAG: @u1
__device__ int u1;

// DEV-DAG: @u2
__constant__ int u2;

// Check host-used static device var is in llvm.compiler.used.
// DEV-DAG: @_ZL2u3
static __device__ int u3;

// Check device-used static device var is emitted but is not in llvm.compiler.used.
// DEV-DAG: @_ZL2u4
static __device__ int u4;

// Check device variables with used attribute are always kept.
// DEV-DAG: @u5
__device__ __attribute__((used)) int u5;

// Test external device variable ODR-used by host code is not emitted or registered.
// DEV-NEG-NOT: @ext_var
extern __device__ int ext_var;

// DEV-DAG: @inline_var = linkonce_odr addrspace(1) externally_initialized global i32 0
__device__ inline int inline_var;

template<typename T>
using func_t = T (*) (T, T);

template <typename T>
__device__ T add_func (T x, T y)
{
  return x + y;
}

// DEV-DAG: @_Z10p_add_funcIiE = linkonce_odr addrspace(1) externally_initialized global i32 (i32, i32)* @_Z8add_funcIiET_S0_S0_
template <typename T>
__device__ func_t<T> p_add_func = add_func<T>;

// Check non-constant constexpr variables ODR-used by host code only is not emitted.
// DEV-NEG-NOT: constexpr_var1a
// DEV-NEG-NOT: constexpr_var1b
constexpr int constexpr_var1a = 1;
inline constexpr int constexpr_var1b = 1;

// Check constant constexpr variables ODR-used by host code only.
// Non-inline constexpr variable has internal linkage, therefore it is not accessible by host and not kept.
// Inline constexpr variable has linkonce_ord linkage, therefore it can be accessed by host and kept.
// DEV-NEG-NOT: constexpr_var2a
// DEV-DAG: @constexpr_var2b = linkonce_odr addrspace(4) externally_initialized constant i32 2
__constant__ constexpr int constexpr_var2a = 2;
inline __constant__ constexpr int constexpr_var2b = 2;

void use(func_t<int> p);
void use(const int *p);

void fun1() {
  use(&u1);
  use(&u2);
  use(&u3);
  use(&ext_var);
  use(&inline_var);
  use(p_add_func<int>);
  use(&constexpr_var1a);
  use(&constexpr_var1b);
  use(&constexpr_var2a);
  use(&constexpr_var2b);
}

__global__ void kern1(int **x) {
  *x = &u4;
}

// Check implicit constant variable ODR-used by host code is not emitted.
// DEV-NEG-NOT: _ZN16TestConstexprVar1oE
namespace TestConstexprVar {
char o;
class ou {
public:
  ou(char) { __builtin_strlen(&o); }
};
template < typename ao > struct aw { static constexpr ao c; };
class x {
protected:
  typedef ou (*y)(const x *);
  constexpr x(y ag) : ah(ag) {}
  template < bool * > struct ak;
  template < typename > struct al {
    static bool am;
    static ak< &am > an;
  };
  template < typename ao > static x ap() { (void)aw< ao >::c; return x(nullptr); }
  y ah;
};
template < typename ao > bool x::al< ao >::am(&ap< ao >);
class ar : x {
public:
  constexpr ar() : x(as) {}
  static ou as(const x *) { return 0; }
  al< ar > av;
};
}

// Check the exact list of variables to ensure @_ZL2u4 is not among them.
// DEV: @llvm.compiler.used = {{[^@]*}} @_Z10p_add_funcIiE {{[^@]*}} @_ZL2u3 {{[^@]*}} @constexpr_var2b {{[^@]*}} @inline_var {{[^@]*}} @u1 {{[^@]*}} @u2 {{[^@]*}} @u5

// HOST-DAG: hipRegisterVar{{.*}}@u1
// HOST-DAG: hipRegisterVar{{.*}}@u2
// HOST-DAG: hipRegisterVar{{.*}}@_ZL2u3
// HOST-DAG: hipRegisterVar{{.*}}@constexpr_var2b
// HOST-DAG: hipRegisterVar{{.*}}@u5
// HOST-DAG: hipRegisterVar{{.*}}@inline_var
// HOST-DAG: hipRegisterVar{{.*}}@_Z10p_add_funcIiE
// HOST-NEG-NOT: hipRegisterVar{{.*}}@ext_var
// HOST-NEG-NOT: hipRegisterVar{{.*}}@_ZL2u4
// HOST-NEG-NOT: hipRegisterVar{{.*}}@constexpr_var1a
// HOST-NEG-NOT: hipRegisterVar{{.*}}@constexpr_var1b
// HOST-NEG-NOT: hipRegisterVar{{.*}}@constexpr_var2a
