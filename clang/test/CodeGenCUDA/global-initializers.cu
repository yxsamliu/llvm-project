#include "hip/hip_runtime.h"
// RUN: %clang_cc1 %s -triple x86_64-linux-unknown -emit-llvm -o - \
// RUN:   | FileCheck -check-prefix=HOST %s
// RUN: %clang_cc1 %s -fcuda-is-device \
// RUN:   -emit-llvm -o - -triple nvptx64 \
// RUN:   -aux-triple x86_64-unknown-linux-gnu | FileCheck \
// RUN:   -check-prefix=DEV %s

#include "Inputs/hip/hip_runtime.h"

// Check host/device-based overloding resolution in global variable initializer.
template<typename T, typename U>
T pow(T, U) { return 1.0; }

__device__ double pow(double, int) { return 2.0; }

// HOST-DAG: call {{.*}}double @_Z3powIdiET_S0_T0_(double noundef 1.000000e+00, i32 noundef 1)
double X = pow(1.0, 1);

template<typename T, typename U>
constexpr T cpow(T, U) { return 11.0; }

constexpr __device__ double cpow(double, int) { return 12.0; }

// HOST-DAG: @CX = global double 1.100000e+01
double CX = cpow(11.0, 1);

// DEV-DAG: @CY = addrspace(1) externally_initialized global double 1.200000e+01
__device__ double CY = cpow(12.0, 1);

struct A {
  template<typename T, typename U>
  T pow(T, U) { return 3.0; }

  __device__ double pow(double, int) { return 4.0; }
};

A a;

// HOST-DAG: call {{.*}}double @_ZN1A3powIdiEET_S1_T0_(ptr {{.*}}@a, double noundef 3.000000e+00, i32 noundef 1)
double AX = a.pow(3.0, 1);

struct CA {
  template<typename T, typename U>
  constexpr T cpow(T, U) const { return 13.0; }

  constexpr __device__ double cpow(double, int) const { return 14.0; }
};

const CA ca;

// HOST-DAG: @CAX = global double 1.300000e+01
double CAX = ca.cpow(13.0, 1);

// DEV-DAG: @CAY = addrspace(1) externally_initialized global double 1.400000e+01
__device__ double CAY = ca.cpow(14.0, 1);
