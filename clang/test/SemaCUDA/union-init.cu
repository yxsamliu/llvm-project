#include "hip/hip_runtime.h"
// RUN: %clang_cc1 %s --std=c++11 -triple x86_64-linux-unknown \
// RUN:   -fsyntax-only -o - -verify=com,host
// RUN: %clang_cc1 %s --std=c++11 -triple nvptx -fcuda-is-device \
// RUN:   -fsyntax-only -o - -verify=com,dev

#include "Inputs/hip/hip_runtime.h"

struct A {
  int a;
  __device__ A() { a = 1; }
  __device__ ~A() { a = 2; }
};

// This can be a global var since ctor/dtors of data members are not called.
union B {
  A a;
  __device__ B() {}
  __device__ ~B() {}
};

// This cannot be a global var since it has a dynamic ctor.
union C {
  A a;
  __device__ C() { a.a = 3; }
  __device__ ~C() {}
};

// This cannot be a global var since it has a dynamic dtor.
union D {
  A a;
  __device__ D() { }
  __device__ ~D() { a.a = 4; }
};

__device__ B b;
__device__ C c;
// com-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}
__device__ D d;
// com-error@-1 {{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables.}}

__device__ void foo() {
  __shared__ B b;
  __shared__ C c;
  // com-error@-1 {{initialization is not supported for __shared__ variables.}}
  __shared__ D d;
  // com-error@-1 {{initialization is not supported for __shared__ variables.}}
}
